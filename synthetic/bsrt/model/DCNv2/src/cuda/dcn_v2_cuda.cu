#include "hip/hip_runtime.h"
#include <vector>
#include "cuda/dcn_v2_im2col_cuda.h"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/Dispatch.h>
#include <ATen/div_rtn.h>
#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/Exceptions.h>

THCState *state = at::globalContext().lazyInitCUDA();

static hipblasOperation_t _cublasOpFromChar(char op) {
    switch (op) {
      case 'n':
      case 'N':
        return HIPBLAS_OP_N;
      case 't':
      case 'T':
        return HIPBLAS_OP_T;
      case 'c':
      case 'C':
        return HIPBLAS_OP_C;
    }
    AT_ERROR(
        "_cublasOpFromChar input should be 't', 'n' or 'c' but got `", op, "`");
  }

  static void _cublasAdjustLdLevel2(int64_t m, int64_t n, int64_t* lda) {
    // Note: leading dimensions generally are checked that they are > 0
    // and at least as big the result requires (even if the value won't
    // be used).
  
    // Q: Why does Level3 check trans but this doesn't?
    // A: In level 2, the sizes (m, n) specify the size of A
    // (independent of trans value). In level 3. the sizes (m, n, k)
    // specify the sizes of op(A), op(B) where op depend on trans
    // values.
    if (n <= 1)
      *lda = std::max<int64_t>(m, 1);
  }



// author: Charles Shang
// https://github.com/torch/cunn/blob/master/lib/THCUNN/generic/SpatialConvolutionMM.cu

// [batch gemm]
// https://github.com/pytorch/pytorch/blob/master/aten/src/THC/generic/THCTensorMathBlas.cu

__global__ void createBatchGemmBuffer(const float **input_b, float **output_b,
                                      float **columns_b, const float **ones_b,
                                      const float **weight_b, const float **bias_b,
                                      float *input, float *output,
                                      float *columns, float *ones,
                                      float *weight, float *bias,
                                      const int input_stride, const int output_stride,
                                      const int columns_stride, const int ones_stride,
                                      const int num_batches)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_batches)
    {
        input_b[idx] = input + idx * input_stride;
        output_b[idx] = output + idx * output_stride;
        columns_b[idx] = columns + idx * columns_stride;
        ones_b[idx] = ones + idx * ones_stride;
        // share weights and bias within a Mini-Batch
        weight_b[idx] = weight;
        bias_b[idx] = bias;
    }
}

at::Tensor
dcn_v2_cuda_forward(const at::Tensor &input,
                    const at::Tensor &weight,
                    const at::Tensor &bias,
                    const at::Tensor &offset,
                    const at::Tensor &mask,
                    const int kernel_h,
                    const int kernel_w,
                    const int stride_h,
                    const int stride_w,
                    const int pad_h,
                    const int pad_w,
                    const int dilation_h,
                    const int dilation_w,
                    const int deformable_group)
{
    using scalar_t = float;
    // THCAssertSameGPU(THCudaTensor_checkGPU(state, 5, input, weight, bias, offset, mask));
    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h_ = weight.size(2);
    const int kernel_w_ = weight.size(3);

    // printf("Kernels: %d %d %d %d\n", kernel_h_, kernel_w_, kernel_w, kernel_h);
    // printf("Channels: %d %d\n", channels, channels_kernel);
    // printf("Channels: %d %d\n", channels_out, channels_kernel);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
               "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == channels_kernel,
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    auto ones = at::ones({batch, height_out, width_out}, input.options());
    auto columns = at::empty({batch, channels * kernel_h * kernel_w, 1 * height_out * width_out}, input.options());
    auto output = at::empty({batch, channels_out, height_out, width_out}, input.options());

    // prepare for batch-wise computing, which is significantly faster than instance-wise computing
    // when batch size is large.
    // launch batch threads
    int matrices_size = batch * sizeof(float *);
    auto input_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));
    auto output_b = static_cast<float **>(THCudaMalloc(state, matrices_size));
    auto columns_b = static_cast<float **>(THCudaMalloc(state, matrices_size));
    auto ones_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));
    auto weight_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));
    auto bias_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));

    const int block = 128;
    const int grid = (batch + block - 1) / block;

    createBatchGemmBuffer<<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
        input_b, output_b,
        columns_b, ones_b,
        weight_b, bias_b,
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        columns.data_ptr<scalar_t>(),
        ones.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        bias.data_ptr<scalar_t>(),
        channels * width * height,
        channels_out * width_out * height_out,
        channels * kernel_h * kernel_w * height_out * width_out,
        height_out * width_out,
        batch);

    long m_ = channels_out;
    long n_ = height_out * width_out;
    long k_ = 1;
    THCudaBlas_SgemmBatched(state,
                            't',
                            'n',
                            n_,
                            m_,
                            k_,
                            1.0f,
                            ones_b, k_,
                            bias_b, k_,
                            0.0f,
                            output_b, n_,
                            batch);

    modulated_deformable_im2col_cuda(c10::cuda::getCurrentCUDAStream(),
                                     input.data_ptr<scalar_t>(),
                                     offset.data_ptr<scalar_t>(),
                                     mask.data_ptr<scalar_t>(),
                                     batch, channels, height, width,
                                     height_out, width_out, kernel_h, kernel_w,
                                     pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
                                     deformable_group,
                                     columns.data_ptr<scalar_t>());

    long m = channels_out;
    long n = height_out * width_out;
    long k = channels * kernel_h * kernel_w;
    THCudaBlas_SgemmBatched(state,
                            'n',
                            'n',
                            n,
                            m,
                            k,
                            1.0f,
                            (const float **)columns_b, n,
                            weight_b, k,
                            1.0f,
                            output_b, n,
                            batch);

    THCudaFree(state, input_b);
    THCudaFree(state, output_b);
    THCudaFree(state, columns_b);
    THCudaFree(state, ones_b);
    THCudaFree(state, weight_b);
    THCudaFree(state, bias_b);
    return output;
}

__global__ void createBatchGemmBufferBackward(
    float **grad_output_b,
    float **columns_b,
    float **ones_b,
    float **weight_b,
    float **grad_weight_b,
    float **grad_bias_b,
    float *grad_output,
    float *columns,
    float *ones,
    float *weight,
    float *grad_weight,
    float *grad_bias,
    const int grad_output_stride,
    const int columns_stride,
    const int ones_stride,
    const int num_batches)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_batches)
    {
        grad_output_b[idx] = grad_output + idx * grad_output_stride;
        columns_b[idx] = columns + idx * columns_stride;
        ones_b[idx] = ones + idx * ones_stride;

        // share weights and bias within a Mini-Batch
        weight_b[idx] = weight;
        grad_weight_b[idx] = grad_weight;
        grad_bias_b[idx] = grad_bias;
    }
}

std::vector<at::Tensor> dcn_v2_cuda_backward(const at::Tensor &input,
                                             const at::Tensor &weight,
                                             const at::Tensor &bias,
                                             const at::Tensor &offset,
                                             const at::Tensor &mask,
                                             const at::Tensor &grad_output,
                                             int kernel_h, int kernel_w,
                                             int stride_h, int stride_w,
                                             int pad_h, int pad_w,
                                             int dilation_h, int dilation_w,
                                             int deformable_group)
{

    THArgCheck(input.is_contiguous(), 1, "input tensor has to be contiguous");
    THArgCheck(weight.is_contiguous(), 2, "weight tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h_ = weight.size(2);
    const int kernel_w_ = weight.size(3);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
               "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == channels_kernel,
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    auto ones = at::ones({height_out, width_out}, input.options());
    auto columns = at::empty({channels * kernel_h * kernel_w, 1 * height_out * width_out}, input.options());
    auto output = at::empty({batch, channels_out, height_out, width_out}, input.options());

    auto grad_input = at::zeros_like(input);
    auto grad_weight = at::zeros_like(weight);
    auto grad_bias = at::zeros_like(bias);
    auto grad_offset = at::zeros_like(offset);
    auto grad_mask = at::zeros_like(mask);

    using scalar_t = float;

    for (int b = 0; b < batch; b++)
    {
        auto input_n = input.select(0, b);
        auto offset_n = offset.select(0, b);
        auto mask_n = mask.select(0, b);
        auto grad_output_n = grad_output.select(0, b);
        auto grad_input_n = grad_input.select(0, b);
        auto grad_offset_n = grad_offset.select(0, b);
        auto grad_mask_n = grad_mask.select(0, b);

        long m = channels * kernel_h * kernel_w;
        long n = height_out * width_out;
        long k = channels_out;

        THCudaBlas_Sgemm(state, 'n', 't', n, m, k, 1.0f,
                         grad_output_n.data_ptr<scalar_t>(), n,
                         weight.data_ptr<scalar_t>(), m, 0.0f,
                         columns.data_ptr<scalar_t>(), n);

        // gradient w.r.t. input coordinate data
        modulated_deformable_col2im_coord_cuda(c10::cuda::getCurrentCUDAStream(),
                                               columns.data_ptr<scalar_t>(),
                                               input_n.data_ptr<scalar_t>(),
                                               offset_n.data_ptr<scalar_t>(),
                                               mask_n.data_ptr<scalar_t>(),
                                               1, channels, height, width,
                                               height_out, width_out, kernel_h, kernel_w,
                                               pad_h, pad_w, stride_h, stride_w,
                                               dilation_h, dilation_w, deformable_group,
                                               grad_offset_n.data_ptr<scalar_t>(),
                                               grad_mask_n.data_ptr<scalar_t>());
        // gradient w.r.t. input data
        modulated_deformable_col2im_cuda(c10::cuda::getCurrentCUDAStream(),
                                         columns.data_ptr<scalar_t>(),
                                         offset_n.data_ptr<scalar_t>(),
                                         mask_n.data_ptr<scalar_t>(),
                                         1, channels, height, width,
                                         height_out, width_out, kernel_h, kernel_w,
                                         pad_h, pad_w, stride_h, stride_w,
                                         dilation_h, dilation_w, deformable_group,
                                         grad_input_n.data_ptr<scalar_t>());

        // gradient w.r.t. weight, dWeight should accumulate across the batch and group
        modulated_deformable_im2col_cuda(c10::cuda::getCurrentCUDAStream(),
                                         input_n.data_ptr<scalar_t>(),
                                         offset_n.data_ptr<scalar_t>(),
                                         mask_n.data_ptr<scalar_t>(),
                                         1, channels, height, width,
                                         height_out, width_out, kernel_h, kernel_w,
                                         pad_h, pad_w, stride_h, stride_w,
                                         dilation_h, dilation_w, deformable_group,
                                         columns.data_ptr<scalar_t>());

        long m_ = channels_out;
        long n_ = channels * kernel_h * kernel_w;
        long k_ = height_out * width_out;

        THCudaBlas_Sgemm(state, 't', 'n', n_, m_, k_, 1.0f,
                         columns.data_ptr<scalar_t>(), k_,
                         grad_output_n.data_ptr<scalar_t>(), k_, 1.0f,
                         grad_weight.data_ptr<scalar_t>(), n_);

        hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
        hipblasOperation_t op = _cublasOpFromChar('t');
        _cublasAdjustLdLevel2(k_, m_, &k_);
        scalar_t* grad_output_n_float = grad_output_n.data_ptr<scalar_t>();
        scalar_t* one_float = ones.data_ptr<scalar_t>();
        scalar_t alpha = 1.0;
        scalar_t beta = 1.0;
        hipblasSgemv(handle, op, k_, m_, &alpha, grad_output_n_float,k_, one_float,1, &beta, grad_bias.data_ptr<scalar_t>(), 1);

    }
    

    return {
        grad_input, grad_offset, grad_mask, grad_weight, grad_bias
    };
}
